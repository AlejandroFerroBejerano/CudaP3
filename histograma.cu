#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <>

#include <image.h>  
#define Tamaño IMAGE_WIDTH*IMAGE_HEIGTH;
#define RangoColores 256
#define Nbloques 1
#define NThreads 256

__global__ void histograma_kernel(unsigned char *buffer, long size, unsigned int *hist){
	
		/*Buffer de histograma temporal en memoria compartida*/	
	__shared__ unsigned int temp[256];
	temp[threadIdx.x]=0;
	__syncthreads();

	int posicion = threadIdx.x + blockIdx.x * blockDim.x;
	int desplazamiento = blockDim.x * gridDim.x;

	while(posicion < size){
		/*Bloquea la variable de memoria compartida para que no escriban en la misma */		
		atomicAdd(&temp[buffer[posicion]], 1);
		posicion += desplazamiento;
	}
	
	/*Esperamos a que todos lo hilos hayan terminado */
	__syncthreads();
	/*Copiamos nuestro histograma en memoria compartida*/
	atomicAdd( &(hist[threadIdx.x]), temp[treadIdx.x]);
}

int main(void){
	unsigned char *image =(unsigned char*)image;
	unsigned int histograma[RangoColores];
	unsigned char *dev_image;
	unsigned int *dev_histograma;

	long = tamaño IMAGE_WIDTH*IMAGE_HEIGTH;

	hipMalloc((void**) &dev_image, tamaño);
	hipMalloc((void**) &dev_histograma, RangoColores * sizeof(long));

	hipMemcpy(dev_image, image, Tamaño, hipMemcpyHostToDevice);
	histograma_kernel<<<Nbloques,NThreads>>>(dev_image,tamaño,dev_histograma);
	hipMemcpy(histograma, &dev_histograma, RangoColores * sizeof(int), hipMemcpyDeviceToHost);

	for (int i=0; i<tamaño; i++){
		printf("%d\t"histograma[i]);	
	}
}
