#include "hip/hip_runtime.h"
/*autor: Alejandro Ferro Bejerano*/

#include <stdio.h>
#include "image.h"

#define TRUE 1
#define FALSE 0
#define SIZE IMAGE_WIDTH * IMAGE_HEIGHT
#define RangoColores 256
#define Nbloques 1 
#define NThreads 256

__global__ void histograma_kernel(unsigned char *img, unsigned int *hist){
	
	/*Inicializamos histograma temporal en memoria compartida a 0*/	
	__shared__ unsigned int temp[RangoColores];
	temp[threadIdx.x]=0;
	__syncthreads();

	int posicion = threadIdx.x + blockIdx.x * blockDim.x;
	int desplazamiento = blockDim.x * gridDim.x;

	while(posicion < SIZE){
		/*Bloquea la variable de memoria compartida para que no escriban en la misma */		
		atomicAdd(&temp[img[posicion]], 1);
		posicion +=desplazamiento;
	}
	
	/*Esperamos a que todos lo hilos hayan terminado */
	__syncthreads();
	/*Copiamos de nuestra memoria compartida a nuestro histograma*/
	atomicAdd( &(hist[threadIdx.x]), temp[threadIdx.x]);
}

int main(void){

	int hist_correcto = FALSE;
	/*Cargamos la imagen*/
	unsigned char *img =(unsigned char*)image;

	/*Declaramos el array histograma y los punteros a la imagen y al histograma en memoria*/
	unsigned int histograma[RangoColores];
	unsigned char *dev_image;
	unsigned int *dev_histograma;


	/*Reservamos memoria e inicializamos a 0 
	todo el rango donde se almacenara el histograma*/
	hipMalloc((void**) &dev_image, SIZE);
	hipMalloc((void**) &dev_histograma, RangoColores * sizeof(int));
	hipMemset( dev_histograma, 0,RangoColores * sizeof( int ) );

	hipMemcpy(dev_image, img, SIZE, hipMemcpyHostToDevice);
	histograma_kernel<<<Nbloques,NThreads>>>(dev_image,dev_histograma);
	hipMemcpy(histograma, dev_histograma, RangoColores * sizeof(int), hipMemcpyDeviceToHost);
	
		/*Comprobamos uqe el cálculo se ha hecho correctamente haciendo
	la operación inversa con la CPU*/
	for (int i=0; i<SIZE; i++) histograma[img[i]]--;
	for(int i=0; i< RangoColores; i++){
		if (histograma[i] !=0){
			printf("\nError: El cálculo del histograma, no corresponde con el generado por la CPU\n\n");
			hist_correcto = FALSE;
			exit(-1);
		}else{
			hist_correcto = TRUE;
		}
	}
	if(hist_correcto == TRUE) printf("Histograma generado correctamente, ;-)\n\n");

	
	hipFree(dev_image);
	hipFree(dev_histograma);
	
	return 0;
}
